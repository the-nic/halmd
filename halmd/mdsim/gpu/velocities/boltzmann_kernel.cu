#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2010  Peter Colberg and Felix Höfling
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/algorithm/gpu/reduction.cuh>
#include <halmd/mdsim/gpu/velocities/boltzmann_kernel.hpp>
#include <halmd/random/gpu/normal_distribution.cuh>
#include <halmd/random/gpu/random_number_generator.cuh>
#include <halmd/utility/gpu/thread.cuh>

using namespace halmd::algorithm::gpu;

//
// Maxwell-Boltzmann distribution at accurate temperature
//

namespace halmd {
namespace mdsim {
namespace gpu {
namespace velocities {
namespace boltzmann_kernel {

/**
 * generate Maxwell-Boltzmann distributed velocities and reduce velocity
 */
template <
    typename vector_type
  , typename rng_type
  , int threads
  , typename T
>
__global__ void gaussian(
    float4* g_v
  , unsigned int const* g_group
  , unsigned int npart
  , unsigned int nplace
  , float temp
  , T* g_mv
  , dsfloat* g_mv2
  , dsfloat* g_m
  , rng_type rng
)
{
    enum { dimension = vector_type::static_size };
    typedef typename vector_type::value_type float_type;

    extern __shared__ char __s_array[];
    fixed_vector<dsfloat, dimension>* const s_mv = reinterpret_cast<fixed_vector<dsfloat, dimension>*>(__s_array);
    dsfloat* const s_mv2 = reinterpret_cast<dsfloat*>(&s_mv[TDIM]);
    dsfloat* const s_m = reinterpret_cast<dsfloat*>(&s_mv2[TDIM]);

    fixed_vector<dsfloat, dimension> mv = 0;
    dsfloat mv2 = 0;
    dsfloat m = 0;

    // read random number generator state from global device memory
    typename rng_type::state_type state = rng[GTID];

    // normal distribution parameters
    float const mean = 0.f;
    float const sigma = sqrtf(temp);

    // cache second normal variate for uneven dimensions
    bool cached = false;
    typename vector_type::value_type cache;

    for (uint i = GTID; i < npart; i += GTDIM) {
        unsigned int const idx = g_group[i];
        vector_type v;
        float mass;
#ifdef USE_VERLET_DSFUN
        tie(v, mass) <<= tie(g_v[idx], g_v[idx + nplace]);
#else
        tie(v, mass) <<= g_v[idx];
#endif
        for (uint j = 0; j < dimension - 1; j += 2) {
            tie(v[j], v[j + 1]) = normal(rng, state, mean, sigma);
        }
        if (dimension % 2) {
           if ((cached = !cached)) {
               tie(v[dimension - 1], cache) = normal(rng, state, mean, sigma);
           }
           else {
               v[dimension - 1] = cache;
           }
        }
        v /= sqrtf(mass);
        mv += mass * v;
        mv2 += mass * inner_prod(v, v);
        m += mass;
#ifdef USE_VERLET_DSFUN
        tie(g_v[idx], g_v[idx + nplace]) <<= tie(v, mass);
#else
        g_v[id] <<= tie(v, mass);
#endif
    }

    // store random number generator state in global device memory
    rng[GTID] = state;

    // reduced values for this thread
    s_mv[TID] = mv;
    s_mv2[TID] = mv2;
    s_m[TID] = m;
    __syncthreads();

    // compute reduced value for all threads in block
    reduce<threads / 2, ternary_sum_>(mv, mv2, m, s_mv, s_mv2, s_m);

    if (TID < 1) {
        // store block reduced value in global memory
        tie(g_mv[blockIdx.x], g_mv[blockIdx.x + BDIM]) = split(mv);
        g_mv2[blockIdx.x] = mv2;
        g_m[blockIdx.x] = m;
    }
}

template <
    typename vector_type
  , typename T
>
__global__ void shift_rescale(
    float4* g_v
  , unsigned int const* g_group
  , uint npart
  , uint nplace
  , dsfloat temp
  , T const* g_mv
  , dsfloat const* g_mv2
  , dsfloat const* g_m
  , uint size
)
{
    enum { dimension = vector_type::static_size };
    typedef typename vector_type::value_type float_type;

    extern __shared__ char __s_array[];
    fixed_vector<dsfloat, dimension>* const s_mv = reinterpret_cast<fixed_vector<dsfloat, dimension>*>(__s_array);
    dsfloat* const s_mv2 = reinterpret_cast<dsfloat*>(&s_mv[size]);
    dsfloat* const s_m = reinterpret_cast<dsfloat*>(&s_mv2[size]);

    fixed_vector<dsfloat, dimension> mv = 0;
    dsfloat mv2 = 0;
    dsfloat m = 0;

    for (uint i = TID; i < size; i += TDIM) {
#ifdef USE_VERLET_DSFUN
        s_mv[i] = vector_type(g_mv[i], g_mv[i + size]);
#else
        s_mv[i] = vector_type(g_mv[i]);
#endif
        s_mv2[i] = g_mv2[i];
        s_m[i] = g_m[i];
    }
    __syncthreads();
    for (uint i = 0; i < size; ++i) {
        mv += s_mv[i];
        mv2 += s_mv2[i];
        m += s_m[i];
    }

    vector_type vcm = vector_type(mv / m);
    float_type scale = sqrt(npart * temp * static_cast<int>(dimension) / (mv2 - m * inner_prod(vcm, vcm)));

    for (uint i = GTID; i < npart; i += GTDIM) {
        unsigned int const idx = g_group[i];
        vector_type v;
        float mass;
#ifdef USE_VERLET_DSFUN
        tie(v, mass) <<= tie(g_v[idx], g_v[idx + nplace]);
#else
        tie(v, mass) <<= g_v[idx];
#endif
        v -= vcm;
        v *= scale;
#ifdef USE_VERLET_DSFUN
        tie(g_v[idx], g_v[idx + nplace]) <<= tie(v, mass);
#else
        g_v[idx] <<= tie(v, mass);
#endif
    }
}

} // namespace boltzmann_kernel

template <int dimension, typename float_type, typename rng_type>
boltzmann_wrapper<dimension, float_type, rng_type> const boltzmann_wrapper<dimension, float_type, rng_type>::kernel = {
    boltzmann_kernel::gaussian<fixed_vector<float_type, dimension>, rng_type, 32>
  , boltzmann_kernel::gaussian<fixed_vector<float_type, dimension>, rng_type, 64>
  , boltzmann_kernel::gaussian<fixed_vector<float_type, dimension>, rng_type, 128>
  , boltzmann_kernel::gaussian<fixed_vector<float_type, dimension>, rng_type, 256>
  , boltzmann_kernel::gaussian<fixed_vector<float_type, dimension>, rng_type, 512>
  , boltzmann_kernel::shift_rescale<fixed_vector<float_type, dimension> >
};

#ifdef USE_VERLET_DSFUN
template class boltzmann_wrapper<3, dsfloat, random::gpu::rand48_rng>;
template class boltzmann_wrapper<2, dsfloat, random::gpu::rand48_rng>;
#else
template class boltzmann_wrapper<3, float, random::gpu::rand48_rng>;
template class boltzmann_wrapper<2, float, random::gpu::rand48_rng>;
#endif

} // namespace mdsim
} // namespace gpu
} // namespace velocities
} // namespace halmd
