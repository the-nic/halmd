#include "hip/hip_runtime.h"
/*
 * Copyright © 2014 Nicolas Höft
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/region_kernel.hpp>
#include <halmd/mdsim/gpu/box_kernel.cuh>
#include <halmd/utility/gpu/thread.cuh>

#include <halmd/mdsim/geometries/cuboid.hpp>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace region_kernel {

template <typename vector_type, typename geometry_type>
__global__ void compute_mask(
    float4 const* g_r
  , unsigned int nparticle
  , unsigned int* g_mask
  , geometry_type const geometry
  , vector_type box_length
)
{
    enum { dimension = vector_type::static_size };
    unsigned int const i = GTID;
    if(i >= nparticle)
        return;

    vector_type r;
    unsigned int type;
    tie(r, type) <<= g_r[i];

    // enforce periodic boundary conditions
    box_kernel::reduce_periodic(r, box_length);
    // 1 means the particle in in the selector, 0 means outside
    g_mask[i] = geometry(r) ? 1 : 0;
}

/**
 * generate ascending index sequence
 */
__global__ void gen_index(unsigned int* g_index, unsigned int nparticle)
{
    g_index[GTID] = (GTID < nparticle) ? GTID : 0;
}

__global__ void compute_bin_border(
    unsigned int* g_offset
  , unsigned int* const g_mask
  , unsigned int nparticle
)
{
    unsigned int const i = GTID;
    if(i >= nparticle-1)
        return;
    if (g_mask[i+1] > g_mask[i]) {
        *g_offset = i+1;
    }
}

} // namespace region_kernel

template<int dimension, typename geometry_type>
region_wrapper<dimension, geometry_type> const
region_wrapper<dimension, geometry_type>::kernel = {
    region_kernel::compute_mask
  , region_kernel::gen_index
  , region_kernel::compute_bin_border
};

template class region_wrapper<3, halmd::mdsim::geometries::cuboid<3, float> >;
template class region_wrapper<2, halmd::mdsim::geometries::cuboid<2, float> >;

} // namespace gpu
} // namespace mdsim
} // namespace halmd
