#include "hip/hip_runtime.h"
/*
 * Copyright © 2008-2012  Peter Colberg
 *
 * This file is part of HALMD.
 *
 * HALMD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <halmd/mdsim/gpu/box_kernel.cuh>
#include <halmd/mdsim/gpu/integrators/verlet_kernel.hpp>
#include <halmd/numeric/blas/blas.hpp>
#include <halmd/numeric/mp/dsfloat.hpp>
#include <halmd/utility/gpu/thread.cuh>

namespace halmd {
namespace mdsim {
namespace gpu {
namespace integrators {
namespace verlet_kernel {

/**
 * First leapfrog half-step of velocity-Verlet algorithm
 */
template <int dimension, typename float_type, typename gpu_vector_type>
__global__ void integrate(
    float4* g_position
  , gpu_vector_type* g_image
  , float4* g_velocity
  , gpu_vector_type const* g_force
  , unsigned int const* g_group
  , unsigned int nparticle
  , unsigned int nthread
  , float timestep
  , fixed_vector<float, dimension> box_length
)
{
    if (GTID < nparticle) {
        // kernel execution parameters
        unsigned int const thread = g_group[GTID];

        // read position, species, velocity, mass, image, force from global memory
        fixed_vector<float_type, dimension> r, v;
        unsigned int species;
        float mass;
#ifdef USE_VERLET_DSFUN
        tie(r, species) <<= tie(g_position[thread], g_position[thread + nthread]);
        tie(v, mass) <<= tie(g_velocity[thread], g_velocity[thread + nthread]);
#else
        tie(r, species) <<= g_position[thread];
        tie(v, mass) <<= g_velocity[thread];
#endif
        fixed_vector<float, dimension> image = g_image[thread];
        fixed_vector<float, dimension> f = g_force[thread];

        // advance position by full step, velocity by half step
        v += f * (timestep / 2) / mass;
        r += v * timestep;
        image += box_kernel::reduce_periodic(r, box_length);

        // store position, species, velocity, mass, image in global memory
#ifdef USE_VERLET_DSFUN
        tie(g_position[thread], g_position[thread + nthread]) <<= tie(r, species);
        tie(g_velocity[thread], g_velocity[thread + nthread]) <<= tie(v, mass);
#else
        g_position[thread] <<= tie(r, species);
        g_velocity[thread] <<= tie(v, mass);
#endif
        g_image[thread] = image;
    }
}

/**
 * Second leapfrog half-step of velocity-Verlet algorithm
 */
template <int dimension, typename float_type, typename gpu_vector_type>
__global__ void finalize(
    float4* g_velocity
  , gpu_vector_type const* g_force
  , unsigned int const* g_group
  , unsigned int nparticle
  , unsigned int nthread
  , float timestep
)
{
    if (GTID < nparticle) {
        // kernel execution parameters
        unsigned int const thread = g_group[GTID];

        // read velocity, mass, force from global memory
        fixed_vector<float_type, dimension> v;
        float mass;
#ifdef USE_VERLET_DSFUN
        tie(v, mass) <<= tie(g_velocity[thread], g_velocity[thread + nthread]);
#else
        tie(v, mass) <<= g_velocity[thread];
#endif
        fixed_vector<float, dimension> f = g_force[thread];

        // advance velocity by half step
        v += f * (timestep / 2) / mass;

        // store velocity, mass in global memory
#ifdef USE_VERLET_DSFUN
        tie(g_velocity[thread], g_velocity[thread + nthread]) <<= tie(v, mass);
#else
        g_velocity[thread] <<= tie(v, mass);
#endif
    }
}

} // namespace verlet_kernel

template <int dimension>
verlet_wrapper<dimension> const verlet_wrapper<dimension>::wrapper = {
#ifdef USE_VERLET_DSFUN
    verlet_kernel::integrate<dimension, dsfloat>
  , verlet_kernel::finalize<dimension, dsfloat>
#else
    verlet_kernel::integrate<dimension, float>
  , verlet_kernel::finalize<dimension, float>
#endif
};

template class verlet_wrapper<3>;
template class verlet_wrapper<2>;

} // namespace mdsim
} // namespace gpu
} // namespace integrators
} // namespace halmd
